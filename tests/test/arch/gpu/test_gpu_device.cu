#include "hip/hip_runtime.h"
/*************************************************************************************/
/*      Copyright 2009 Barcelona Supercomputing Center                               */
/*                                                                                   */
/*      This file is part of the NANOS++ library.                                    */
/*                                                                                   */
/*      NANOS++ is free software: you can redistribute it and/or modify              */
/*      it under the terms of the GNU Lesser General Public License as published by  */
/*      the Free Software Foundation, either version 3 of the License, or            */
/*      (at your option) any later version.                                          */
/*                                                                                   */
/*      NANOS++ is distributed in the hope that it will be useful,                   */
/*      but WITHOUT ANY WARRANTY; without even the implied warranty of               */
/*      MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                */
/*      GNU Lesser General Public License for more details.                          */
/*                                                                                   */
/*      You should have received a copy of the GNU Lesser General Public License     */
/*      along with NANOS++.  If not, see <http://www.gnu.org/licenses/>.             */
/*************************************************************************************/

/*
<testinfo>
test_generator=gens/gpu-generator
test_CXX=nvcc
</testinfo>
*/

#define __aligned__ ignored
#include "gpudevice.hpp"
#include "gpuprocessor.hpp"
#undef __aligned__
#include <iostream>
#include <stdlib.h>
#include <string.h>


using namespace std;

using namespace nanos;
using namespace nanos::ext;


// Arguments' struct
typedef struct {
   int err;
   int n;
   int * Ah;
   int * Ad;
   int * Bh;
   int * Bd;
} test_args;


// GPU kernels
__global__ void check_host_to_device ( int * a , int * err );
__global__ void check_device_to_device ( int * a, int * err );


// GPU tasks
///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_init ( void * args );
///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_host_to_device ( void * args );
///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_device_to_device ( void * args );
///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_device_to_host ( void * args );
///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_cleanup ( void * args );


int main ( int argc, char **argv )
{
   std::cout << "Testing GPU memory manager ( GPUDevice ): allocate, free, copy in / out / local" << std::endl;

   int i, ngpus = 2, n = 512;
   test_args ** args = ( test_args ** ) malloc( ngpus * sizeof( test_args * ) );
   
   for ( i = 0; i < ngpus; i++ ) {
      args[i] = new test_args();
      args[i]->err = 0;
      args[i]->n = n;
      args[i]->Ah = 0;
      args[i]->Ad = 0;
      args[i]->Bh = 0;
      args[i]->Bd = 0;
   }

   nanos::WG *wg = nanos::myThread->getCurrentWD();

   // Initialization
   for ( i = 0; i < ngpus; i++ ) {
///      test_init( args[i] );

      nanos::WD * wd = new nanos::WD( new nanos::ext::GPUDD( test_init ), sizeof( test_args ) * ngpus, args );
      wd->tied();
      wg->addWork( *wd );
      nanos::sys.submit( *wd );
      
      usleep(500);
   }
   wg->waitCompletion();
   
   for ( i = 0; i < ngpus; i++ ) {
      if ( args[i]->err != 0 ) {
         std::cout << "   [" << i << "] " << args[i]->err << " errors found: data allocation into device memory did not succeed.   FAIL" << std::endl;
         exit( 1 );
      }
   }
   
   std::cout << "   Initialization ... ok!" << std::endl;
   
   // First copy: host to device
   for ( i = 0; i < ngpus; i++ ) {
///      test_host_to_device( args[i] );

      nanos::WD * wd = new nanos::WD( new nanos::ext::GPUDD( test_host_to_device ), sizeof( test_args ) * ngpus, args );
      wd->tied();
      wg->addWork( *wd );
      nanos::sys.submit( *wd );
      
      usleep(500);
   }
   wg->waitCompletion();
   
   for ( i = 0; i < ngpus; i++ ) {
      if ( args[i]->err != 0 ) {
         std::cout << "   [" << i << "] " << args[i]->err << " errors found: copying memory from host to device did not succeed.   FAIL" << std::endl;
         exit( 1 );
      }
   }
   
   std::cout << "   Host --> Device ... ok!" << std::endl;
   
   // Second copy: device to device (local)
   for ( i = 0; i < ngpus; i++ ) {
///      test_device_to_device( args[i] );
      
      nanos::WD * wd = new nanos::WD( new nanos::ext::GPUDD( test_device_to_device ), sizeof( test_args ) * ngpus, args );
      wd->tied();
      wg->addWork( *wd );
      nanos::sys.submit( *wd );
      
      usleep(500);
   }
   wg->waitCompletion();
   
   for ( i = 0; i < ngpus; i++ ) {
      if ( args[i]->err != 0 ) {
         std::cout << "   [" << i << "] " << args[i]->err << " errors found: copying memory from device to device did not succeed.   FAIL" << std::endl;
         exit( 1 );
      }
   }
   
   std::cout << "   Device --> Device ... ok!" << std::endl;

   // Third copy: device to host
   for ( i = 0; i < ngpus; i++ ) {
///      test_device_to_host( args[i] );
      
      nanos::WD * wd = new nanos::WD( new nanos::ext::GPUDD( test_device_to_host ), sizeof( test_args ) * ngpus, args );
      wd->tied();
      wg->addWork( *wd );
      nanos::sys.submit( *wd );

      usleep(500);
   }
   wg->waitCompletion();
   
   for ( i = 0; i < ngpus; i++ ) {
      if ( args[i]->err != 0 ) {
         std::cout << "   [" << i << "] " << args[i]->err << " errors found: copying memory from device to host did not succeed.   FAIL" << std::endl;
         exit( 1 );
      }
   }
   
   std::cout << "   Device --> Host ... ok!" << std::endl;
   
   // CLEANUP
   for ( i = 0; i < ngpus; i++ ) {
///      test_cleanup( args[i] );

      nanos::WD * wd = new nanos::WD( new nanos::ext::GPUDD( test_cleanup ), sizeof( test_args ) * ngpus, args );
      wd->tied();
      wg->addWork( *wd );
      nanos::sys.submit( *wd );
      
      usleep(500);
   }
   wg->waitCompletion();
   
   for ( i = 0; i < ngpus; i++ ) {
      if ( args[i]->err != 0 ) {
         std::cout << "   [" << i << "] " << args[i]->err << " errors found: data cleanup from device memory did not succeed.   FAIL" << std::endl;
         exit( 1 );
      }
   }
   
   std::cout << "   Clean up ... ok!" << std::endl;
   
   std::cout << "End testing GPU memory manager ( GPUDevice )" << std::endl;
    
   
   return 0;
}

///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_init ( void * args )
{
   int id = ((nanos::ext::GPUThread *)nanos::myThread)->getGpuDevice();
   
   test_args ** full_args = ( test_args ** ) args;
   test_args * targs = full_args[id];
   
   size_t size = targs->n * sizeof ( int );
   targs->Ah = ( int * ) malloc ( size );
   targs->Bh = ( int * ) malloc ( size );
   targs->Ad = ( int * ) GPUDevice::allocate( size );
   //std::cout << "[" << id << "] Attempting to malloc " << targs->Ad << std::endl;
   targs->Bd = ( int * ) GPUDevice::allocate( size );
   //std::cout << "[" << id << "] Attempting to malloc " << targs->Bd << std::endl;

   targs->err = 0;

   if ( targs->Ah == 0 ) targs->err++;
   if ( targs->Bh == 0 ) targs->err++;
   if ( targs->Ad == 0 ) targs->err++;
   if ( targs->Bd == 0 ) targs->err++;
   
   usleep( 50 );
}

///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_cleanup ( void * args )
{
   int id = ((nanos::ext::GPUThread *)nanos::myThread)->getGpuDevice();
   
   test_args ** full_args = ( test_args ** ) args;
   test_args * targs = full_args[id];
   
   free( targs->Ah );
   free( targs->Bh );
   //std::cout << "[" << id << "] Attempting to free " << targs->Ad << std::endl;
   GPUDevice::free( targs->Ad );
   //std::cout << "[" << id << "] Attempting to free " << targs->Bd << std::endl;
   GPUDevice::free( targs->Bd );
   
   targs->err = 0;
   
   usleep( 50 );
}

///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_host_to_device ( void * args )
{
   int id = ((nanos::ext::GPUThread *)nanos::myThread)->getGpuDevice();
   
   test_args ** full_args = ( test_args ** ) args;
   test_args * targs = full_args[id];
   
   hipStream_t stream = ((nanos::ext::GPUProcessor *) myThread->runningOn())->getGPUProcessorInfo()->getTransferStream();
   
   int i;   
   size_t size = targs->n * sizeof ( int );

   // Initalize arrays
   for ( i = 0; i < targs->n; i++ ) {
      targs->Ah[i] = i;
      targs->Bh[i] = 1;
   }
   hipMemset( targs->Ad, 0, size );
   hipMemset( targs->Bd, 0, size );

   GPUDevice::copyIn( targs->Ad, ( uint64_t ) targs->Ah, size );
   
   hipStreamSynchronize( stream );
   
   // Launch a kernel to check the copy was successful and get the result back from the GPU
   // Arrays Bd and Bh will contain the error checking result
   check_host_to_device <<< 1, targs->n >>> ( targs->Ad, targs->Bd );
   hipMemcpy( targs->Bh, targs->Bd, size, hipMemcpyDeviceToHost );

   targs->err = 0;
   for ( i = 0; i < targs->n; i++ ) {
      if ( targs->Bh[i] ) {
         std::cout << "Error detected at position " << i << ": " << targs->Bh[i] << std::endl;
         targs->err++;
      }
   }
}

///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_device_to_device ( void * args )
{
   int id = ((nanos::ext::GPUThread *)nanos::myThread)->getGpuDevice();
   
   test_args ** full_args = ( test_args ** ) args;
   test_args * targs = full_args[id];
   
   hipStream_t stream = ((nanos::ext::GPUProcessor *) myThread->runningOn())->getGPUProcessorInfo()->getTransferStream();
   
   int i;   
   size_t size = targs->n * sizeof ( int );
   
   // Initalize arrays
   for ( i = 0; i < targs->n; i++ ) {
      targs->Ah[i] = targs->n - i;
      targs->Bh[i] = 1;
   }
   hipMemcpy( targs->Ad, targs->Ah, size, hipMemcpyHostToDevice );
   hipMemset( targs->Bd, 0, size );

   GPUDevice::copyLocal( targs->Bd, targs->Ad, size );
   
   hipStreamSynchronize( stream );
   
   // Launch a kernel to check the copy was successful and get the result back from the GPU
   // Arrays Ad and Bh will contain the error checking result
   check_device_to_device <<< 1, targs->n >>> ( targs->Bd, targs->Ad );
   hipMemcpy( targs->Bh, targs->Ad, size, hipMemcpyDeviceToHost );

   targs->err = 0;
   for ( i = 0; i < targs->n; i++ ) {
      if ( targs->Bh[i] ) {
         std::cout << "Error detected at position " << i << ": " << targs->Bh[i] << std::endl;
         targs->err++;
      }
   }
}

///#pragma omp target device (cuda) copydeps
///#pragma omp task inout (args)
void test_device_to_host ( void * args )
{
   int id = ((nanos::ext::GPUThread *)nanos::myThread)->getGpuDevice();
   
   test_args ** full_args = ( test_args ** ) args;
   test_args * targs = full_args[id];
   
   hipStream_t stream = ((nanos::ext::GPUProcessor *) myThread->runningOn())->getGPUProcessorInfo()->getTransferStream();
   
   int i;   
   size_t size = targs->n * sizeof ( int );
   
   // Initialize arrays
   for ( i = 0; i < targs->n; i++ ) {
      targs->Ah[i] = i;
   }
   memset( targs->Bh, 0, size ); 
   hipMemcpy( targs->Bd, targs->Ah, size, hipMemcpyHostToDevice );

   GPUDevice::copyOut( ( uint64_t ) targs->Bh, targs->Bd, size );

   targs->err = 0;
   for ( i = 0; i < targs->n; i++ ) {
      if ( targs->Bh[i] != targs->Ah[i] ) {
         std::cout << "Error detected at position " << i << ": " << targs->Bh[i] << std::endl;
         targs->err++;
      }
   }
}



/***** GPU CODE *****/
__global__ void check_host_to_device ( int * a , int * err )
{
   int i = threadIdx.x;

   err[i] = a[i] - i;

   a[i] = blockDim.x - threadIdx.x;

}

__global__ void check_device_to_device ( int * b, int * err )
{
   int i = threadIdx.x;

   err[i] = b[i] - (blockDim.x - i);

   b[i] = threadIdx.x;

}
/***** END GPU CODE *****/

